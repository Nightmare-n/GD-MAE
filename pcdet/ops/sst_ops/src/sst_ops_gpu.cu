
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
#define EMPTY_KEY -1

__device__ inline int simple_hash(int k, int hash_size) {
    return k % hash_size;
}

__global__ void ingroup_inds_kernel(int N, const long *group_inds, long *out_inds, int *ingroup_counter) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= N) return;
    long this_group_id = group_inds[pt_idx];
    int cnt = atomicAdd(ingroup_counter + this_group_id, 1);
    out_inds[pt_idx] = cnt;
}

__global__ void group_inner_inds_kernel(int N, int K, const long *inverse_inds, long *group_inds, int *ingroup_counter) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= N) return;
    long this_group_id = inverse_inds[pt_idx];
    int cnt = atomicAdd(ingroup_counter + this_group_id, 1);
    if (cnt < K) group_inds[this_group_id * K + cnt] = pt_idx;
}

__global__ void repeat_group_idx_kernel(int M, int K, const int *ingroup_counter, long *group_inds){
    // params ingroup_counter: (M,)
    // params group_inds: (M, K)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= M) return;
    int cnt = ingroup_counter[pt_idx];
    if (cnt == 0) return;
    for (int i = cnt; i < K; i++) 
        group_inds[pt_idx * K + i] = group_inds[pt_idx * K + i % cnt];
}

void ingroup_inds_launcher(int N, int max_group_id, const long *group_inds, long *out_inds) {
    int *ingroup_counter = NULL;
    hipMalloc(&ingroup_counter, (max_group_id + 1) * sizeof(int));
    hipMemset(ingroup_counter, 0, (max_group_id + 1) * sizeof(int));
    
    dim3 blocks(DIVUP(N, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);

    ingroup_inds_kernel<<<blocks, threads>>>(N, group_inds, out_inds, ingroup_counter);
    hipFree(ingroup_counter);
}

void group_inner_inds_launcher(int N, int M, int K, const long *inverse_inds, long *group_inds) {
    int *ingroup_counter = NULL;
    hipMalloc(&ingroup_counter, M * sizeof(int));
    hipMemset(ingroup_counter, 0, M * sizeof(int));
    
    dim3 blocks(DIVUP(N, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    group_inner_inds_kernel<<<blocks, threads>>>(N, K, inverse_inds, group_inds, ingroup_counter);

    dim3 blocks1(DIVUP(M, THREADS_PER_BLOCK));
    repeat_group_idx_kernel<<<blocks1, threads>>>(M, K, ingroup_counter, group_inds);
    hipFree(ingroup_counter);
}
